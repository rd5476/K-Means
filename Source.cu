#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<vector>
#include<sstream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;


 void display(int * sample, int len){

	for (int jiter = 0; jiter < len; jiter++){
		printf("%f,", sample[jiter]);
	}
	printf("\n");


}

 // Read CSV file to get data , num of rows and num of colums
int ** readCSV(string filename,int * rowNum,int * colNum){
	ifstream ifs("data.csv", ifstream::in);
	if (!ifs){
		printf("Failed");
	}
	vector<vector<int>> dataMat;
	int **data;
	string temp = "ttt";
	while (getline(ifs, temp)){

		if (temp.empty()) continue;
		vector<int> t1;
		istringstream ss(temp);
		string each;
		while (getline(ss, each, ','))
		{

			t1.push_back(stoi(each));
		}
		dataMat.push_back(t1);
		

	}

	int cols = dataMat[0].size();
	*colNum = cols;
	*rowNum = dataMat.size();
	data = new int*[dataMat.size()];
	
	for (int i = 0; i < dataMat.size(); i++){
		data[i] = new int[cols];

	}

	for (int i = 0; i < dataMat.size(); i++){
		for (int jiter = 0; jiter < cols; jiter++){
		data[i][jiter] = dataMat[i][jiter];
		}
	}
	return data;
}


// Kernel to get calculate euclidean distance
//Each thread computes for each data samples
__global__ void KMeans(int rowNum, int colNum,int k, float *centers,int * dataMat,int *label){
	
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	
	if (idx > rowNum) return;
	float min = 999999999;
	
	int curLabel=-1;
	for (int iter=0; iter < k; iter++){
		float dist = 0;
		for (int jiter = 0; jiter < colNum; jiter++){
			dist += (dataMat[idx *colNum + jiter] - centers[iter *colNum + jiter])*(dataMat[idx*colNum + jiter] - centers[iter*colNum + jiter]);
		}
		
		if (dist < min){
			min = dist;
			curLabel = iter;
			
		} 
	}
//	printf("%d - %d - %f\n", idx, curLabel, min);
	label[idx] = curLabel;
}
/*__global__ void UpdateCenter(int rowNum, int colNum, int k, float **centers, int ** dataMat, int *label){
	__shared__ float *centerSum;
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	for (int iter = 0; iter < colNum; iter++){
		int locIdx = label[idx] * colNum + iter;
	}

}*/
int main(){

	
	int rowLen, colNum;
	int**	dataMat = readCSV("data.csv",&rowLen,&colNum);
	
	int k = 4;
	int *clusterSize = new int[k]; 
	
	const unsigned int numThreadsPerClusterBlock = 128;
	const unsigned int numClusterBlocks =(rowLen + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;

	int * dataMat_Dev, *label_Dev;
	float * kCenters_Dev,*kcenters;
	
	kcenters = new float[k*colNum];
	
	//Initial Random K data points
	
	for (int iter = 0; iter < k; iter++){
	
		int idx = rand() % rowLen;
		clusterSize[iter] = 0;
		for (int jiter = 0; jiter < colNum; jiter++){
			kcenters[iter * colNum + jiter] = dataMat[idx][jiter];
		}

	}

	//Display Initial K Centroids
	cout<<"Initial K Centroids\n";
	for (int iter = 0; iter < k; iter++){
		for (int jiter = 0; jiter < colNum; jiter++){
			printf("%f,", kcenters[iter * colNum + jiter]);
		}
		printf("\n");
	}
	

	int delta = 0;


	//Convert 2D matrix to 1D Matrix
	int *flatData = new int[rowLen*colNum];
	for (int iter = 0; iter < rowLen; iter++){
		for (int jiter = 0; jiter < colNum; jiter++){
			flatData[iter * colNum + jiter] = dataMat[iter][jiter];
		}
		
	}
	int *label = new int[rowLen];
	
	// Initialization 
	hipMalloc(&dataMat_Dev, rowLen*colNum*sizeof(int));
	hipMalloc(&kCenters_Dev, k*colNum*sizeof(float));
	hipMalloc(&label_Dev, rowLen*sizeof(int));
	
	memset(label, 0, rowLen);

	hipMemcpy(dataMat_Dev, flatData, rowLen*colNum*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(kCenters_Dev, kcenters, k*colNum*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(label_Dev, label,rowLen*sizeof(int),hipMemcpyHostToDevice);

	
	int *prevLabel = new int[rowLen];
	memset(prevLabel, -1, rowLen);
	int prevdelta = 2* rowLen;
	int change= 0;
	
	
	//Start Iteration : Displays Delta
	cout << "\nDelta\n";
	do{
		delta = 0;
		KMeans << < numClusterBlocks, numThreadsPerClusterBlock >> > (rowLen, colNum, k, kCenters_Dev, dataMat_Dev, label_Dev);
		hipDeviceSynchronize();


		for (int iter = 0; iter < k; iter++){
			clusterSize[iter] = 1;
			for (int jiter = 0; jiter < colNum; jiter++){
				kcenters[iter *colNum+ jiter] = 0;
			}
		}

		hipMemcpy(label, label_Dev, rowLen*sizeof(int), hipMemcpyDeviceToHost);
	
		for (int i = 0; i < rowLen; i++){
			int centerId = label[i];
			if (centerId != prevLabel[i]) delta++;
		
			for (int j = 0; j < colNum; j++){
				kcenters[centerId * colNum + j] += dataMat[i][j];
			}
			clusterSize[centerId]++;
		}

		for (int i = 0; i < k; i++){
			for (int j = 0; j < colNum; j++){
				kcenters[i* colNum + j] /= clusterSize[i];
			}
		}
		hipMemcpy(kCenters_Dev, kcenters, k*colNum*sizeof(float), hipMemcpyHostToDevice); 
		memcpy(prevLabel, label, rowLen);
		
		cout << delta << "\n";
		change = prevdelta - delta;
		prevdelta = delta;
	} while (change>0);
	
	cout << "\nFinal Centroids\n";
	hipMemcpy(kcenters, kCenters_Dev, k*colNum*sizeof(float), hipMemcpyDeviceToHost);
	for (int iter = 0; iter < k; iter++){
		for (int jiter = 0; jiter < colNum; jiter++){
			printf("%f,",kcenters[iter * colNum+ jiter]);
		}
		printf( "\n");
	}
//	hipMemcpy(label, label_Dev, rowLen*sizeof(int), hipMemcpyDeviceToHost);
//	for (int i = 0; i < rowLen; i++){
//		cout << label[i] << "\n";
//	}
	getchar();
	return 0;
}